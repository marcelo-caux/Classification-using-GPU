
#include <hip/hip_runtime.h>
#include <iostream> 
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <stdlib.h>
#include <locale>
#include <string>
#include <limits>
#include <time.h>
#include <stdio.h>
#include <iomanip>
#include <sys/time.h>

using namespace std;

//------------ Kernel de Processamento
__global__ void Classif(int* d_dados, int* d_class, long dsize, int colsIn, int colsOut) 

{
	int i=(threadIdx.x * colsIn) + (blockIdx.x * blockDim.x * colsIn);
	int o=(threadIdx.x * colsOut) + (blockIdx.x * blockDim.x * colsOut);
	int VlOpen,VlHigh,VlLow,VlClose,classe;
	//int classe;

	if (i<=dsize) {
		VlOpen = d_dados[i+1];
		VlHigh = d_dados[i+2];
		VlLow = d_dados[i+3];
		VlClose = d_dados[i+4];
		
		classe=(VlOpen==VlClose ? 512: VlOpen>VlClose ? 256:1024)+(VlLow<VlOpen ? 1:4)+(VlLow<VlClose ? 2:8)+(VlHigh>VlOpen ? 16:64)+(VlHigh>VlClose ? 32:128);
		//classe=(d_dados[i+1]==d_dados[i+4] ? 512: d_dados[i+1]>d_dados[i+4] ? 256:1024)+(d_dados[i+3]<d_dados[i+1] ? 1:4)+(d_dados[i+3]<d_dados[i+4] ? 2:8)+(d_dados[i+2]>d_dados[i+1] ? 16:64)+(d_dados[i+2]>d_dados[i+4] ? 32:128);

		d_class[o]=d_dados[i];
		//d_class[o]=12;
		d_class[o+1]=classe;
	}
}

//--------------------- Funcoes de tempo --------------------------------
std::string DataHora()
{
  time_t rawtime;
  struct tm * timeinfo;
  char buffer [20];

  time ( &rawtime );
  timeinfo = localtime ( &rawtime );
  strftime (buffer,20,"%F %H%M%S",timeinfo);
  return buffer;
}
/* funcao de tempo */
double calcula_tempo(const unsigned long int ini, const unsigned long int fim)
{
  double r;

  if(fim >= ini)
    r = ((double)(fim - ini)) / CLOCKS_PER_SEC;
  else
    r = ((double)( (fim + (unsigned long int)-1) - ini)) / CLOCKS_PER_SEC;
  return r;
}



//------- Classif_paralela:: / std::string ---------------------------

void Classif_GPU(const char * nome, long plins, int nthd, const char * sthd){

	char 			arq[256];
	//char 			arqo[256];
	//std::ifstream 	fin;
	int 			colsIn=5, colsOut=2;
	long 			lins,i, c, last_i_proc, last_c_proc;
	int				dsize, csize, st_dsize, st_csize, partes, st_gatilho;
	//int 			classe,VlOpen,VlHigh,VlLow,VlClose;
	int 			v_blocos,v_threads, streams_processados, d_deslocamento,c_deslocamento;
	std::string		sIndice,sVlOpen,sVlHigh,sVlLow,sVlClose;
	unsigned long int 	t_ini;
	unsigned long int 	t_fin;
	unsigned long int 	t_tmp;
	unsigned long int 	t_tmp1;
	unsigned long int 	t_tmp2;
	unsigned long int 	t_tmp3;
	unsigned long int 	t_tmp4;

	std::string dateStr,fn,fnl,s_threads;
	/*--- define variaveis de tempo -------------*/
	timeval start, end;
	double delta;


	dateStr=DataHora();

	std::cout<<"  <DataHora > = "<<dateStr<<std::endl;

	/* tempo inicial */
  	t_ini  = (unsigned long int) clock;
  	gettimeofday(&start, NULL); //marcador de início do processamento

  	/* -- define as dimensões dos vetores que serão criados em logar de matrizes */
  	/* -- dsize define o tamanho do vetor de dados em função do numero de linhas e colunas*/
  	dsize=plins*colsIn;

   	/* -- csize define o tamanho do vetor de classificacao em função do numero de linhas e colunas*/
  	csize=plins*colsOut;

  	/* ----- Calcula o tamanho dos streams, de acordo com o numero de partes -----------*/
  	partes=40;
  	st_dsize=0;
  	st_csize=0;
  	st_dsize=(int)floor((int)dsize/partes);
  	st_csize=(int)floor((int)csize/partes);

  	/* ----- Calcula o ponto de executar os streams, de acordo com o numero de partes, mas a cada gatilho executa 2 streams -----------*/
  	st_gatilho=(int)floor((int)dsize/partes);
  	st_gatilho*=2;

  	/* -- Cria os vetores que conterão os dados lidos do arquivo e a classificação */
  	int *h_dados;
  	int *h_class;

 	int *d_dados_0;
  	int *d_class_0;
 	int *d_dados_1;
  	int *d_class_1;

  	/*-------------------------- Define os streams ----------------------------------------*/
  	hipStream_t strm0, strm1;
  	hipStreamCreate(&strm0);
   	hipStreamCreate(&strm1);

  	std::cout<<" vai alocar memoria na GPU st_dsize= "<< st_dsize <<" st_csize= "<< st_csize<<std::endl;

 	/*-------------------------- Aloca os vetores no device ----------------------------------------*/
  	hipMalloc((void**) &d_dados_0, st_dsize * sizeof(int));
  	hipMalloc((void**) &d_class_0, st_csize * sizeof(int));
 	hipMalloc((void**) &d_dados_1, st_dsize * sizeof(int));
  	hipMalloc((void**) &d_class_1, st_csize * sizeof(int));

  	/*-------------------------- Aloca os vetores no host ----------------------------------------*/
 	hipHostAlloc((void**) &h_dados, dsize*sizeof(int),hipHostMallocDefault);
 	hipHostAlloc((void**) &h_class, csize*sizeof(int),hipHostMallocDefault);

 	lins=plins-0; 
  	std::cout<<"  <inicializou lns> = "<<lins<<std::endl;

  	/*--- pega o num de threads digitadas e calcula os blocos ------------------------- */
    v_threads=nthd;
    s_threads=std::string(sthd);
    v_blocos=(int)ceil((float)(lins/partes)/v_threads);
    std::cout<<"  <Calculou v_blocos com "<< v_blocos <<" threads com "<< v_threads<<" st_gatilho com "<< st_gatilho <<" dsize="<<dsize<<std::endl;


  	/* -----  Abre o arquivo csv e inicia a carga dos vetores ------------------- */
	strcpy(arq,nome);
	ifstream fin(arq);

    t_tmp1=(unsigned long int) clock();

	if (fin.is_open()) 
	{	  	
	  	t_tmp=(unsigned long int) clock();

	  	/*---  carrega o arquivo no vetor host h_dados e inicializa h_class, transformando valores float em int*/
	  	i=0;
	  	c=0;
	  	streams_processados=0;
	  	c_deslocamento=0;
	  	d_deslocamento=0;
	  	while (fin.good())
	  	{
			getline(fin,sIndice,',');
	      	getline(fin,sVlOpen,',');
	      	getline(fin,sVlHigh,',');
	      	getline(fin,sVlLow,',');
	      	getline(fin,sVlClose,'\n');
	      	//std::cout<<"sIndice= "<< sIndice <<"sVlOpen= "<< sVlOpen<<"sVlHigh= "<< sVlHigh<<"sVlLow= "<< sVlLow<<"sVlClose= "<< sVlClose<<std::endl;
	      	//h_dados[i]=std::stoi(sIndice);
	      	h_dados[i]=std::atoi(sIndice.c_str());
	      	//h_dados[i+1]=static_cast<int>(std::stof(sVlOpen,NULL)*100);
	      	h_dados[i+1]=static_cast<int>(std::atof(sVlOpen.c_str())*100);
	      	h_dados[i+2]=static_cast<int>(std::atof(sVlHigh.c_str())*100);
	      	h_dados[i+3]=static_cast<int>(std::atof(sVlLow.c_str())*100);
	      	h_dados[i+4]=static_cast<int>(std::atof(sVlClose.c_str())*100);

	      	h_class[c]=0;
	      	h_class[c+1]=0;
	      	
	      	//std::cout<<"Indice= "<< h_dados[i] <<"VlOpen= "<< h_dados[i+1]<<"VlHigh= "<< h_dados[i+2]<<"sVlLow= "<< h_dados[i+3]<<"VlClose= "<< h_dados[i+4]<<std::endl;
	  		
	  		/*--- Se atingiu o ponto de transferir os dados (st_gatilho) ou atingiu o último indice de dados -----------
			---- st_dsize-colsOut significa o último registro do stream, st_dsize é o inicio do próximo stream --------
	  		-------------------- copia os vetores e dispara o kernel -------------------------------------------------*/

	      	if ((i>0) && (i<dsize)) {
		  		if ((i % st_gatilho) == 0)
		  		{
		  			c_deslocamento=streams_processados*st_csize;
		  			d_deslocamento=streams_processados*st_dsize;

		  			//std::cout<<"i= "<< i <<" st_dsize= "<< st_dsize<<" d_deslocamento= "<< d_deslocamento<<" c_deslocamento= "<<c_deslocamento<<" streams_processados= "<< streams_processados<<std::endl;

		  			hipMemcpyAsync(d_dados_0,h_dados+d_deslocamento,st_dsize * sizeof(int),hipMemcpyHostToDevice, strm0);
		  			hipMemcpyAsync(d_class_0,h_class+c_deslocamento,st_csize * sizeof(int),hipMemcpyHostToDevice, strm0);
	    			/*--- invoca o kernel de classificação ---*/
	    			Classif<<<v_blocos,v_threads,0, strm0>>>(d_dados_0, d_class_0, st_dsize, colsIn, colsOut);
		  			hipMemcpyAsync(h_class+c_deslocamento,d_class_0,st_csize * sizeof(int),hipMemcpyDeviceToHost, strm0);

	    			streams_processados++;
		  			c_deslocamento=streams_processados*st_csize;
		  			d_deslocamento=streams_processados*st_dsize; 

		  			//std::cout<<"i= "<< i <<" st_dsize= "<< st_dsize<<" d_deslocamento= "<< d_deslocamento<<" c_deslocamento= "<<c_deslocamento<<" streams_processados= "<< streams_processados<<std::endl;		

		  			hipMemcpyAsync(d_dados_1,h_dados+d_deslocamento,st_dsize * sizeof(int),hipMemcpyHostToDevice, strm1);
		  			hipMemcpyAsync(d_class_1,h_class+c_deslocamento,st_csize * sizeof(int),hipMemcpyHostToDevice, strm1);
	    			/*--- invoca o kernel de classificação ---*/
	    			Classif<<<v_blocos,v_threads,0, strm1>>>(d_dados_1, d_class_1, st_dsize, colsIn, colsOut);
		  			hipMemcpyAsync(h_class+c_deslocamento,d_class_1,st_csize * sizeof(int),hipMemcpyDeviceToHost, strm1);

	    			streams_processados++;
	    			last_i_proc=i;
	    			last_c_proc=c;
		  		}
	  		} else {
	  			if (i == dsize) {
		  			c_deslocamento=csize-last_c_proc; //((streams_processados*st_csize)+st_csize);
		  			d_deslocamento=dsize-last_i_proc; //((streams_processados*st_dsize)+st_dsize);

		  			//std::cout<<"i= "<< i <<" st_dsize= "<< st_dsize<<" d_deslocamento= "<< d_deslocamento<<" c_deslocamento= "<<c_deslocamento<<" streams_processados= "<< streams_processados<<std::endl;

		  			hipMemcpyAsync(d_dados_0,h_dados+d_deslocamento,st_dsize * sizeof(int),hipMemcpyHostToDevice, strm0);
		  			hipMemcpyAsync(d_class_0,h_class+c_deslocamento,st_csize * sizeof(int),hipMemcpyHostToDevice, strm0);
	    			/*--- invoca o kernel de classificação ---*/
	    			Classif<<<v_blocos,v_threads,0, strm0>>>(d_dados_0, d_class_0, st_dsize, colsIn, colsOut);
		  			hipMemcpyAsync(h_class+c_deslocamento,d_class_0,st_csize * sizeof(int),hipMemcpyDeviceToHost, strm0);	  				
	  			}
	  		}

	      	i+=colsIn;
	      	c+=colsOut;
	    }

	    std::cout<<"  <Carregou h_dados com "<< i <<" posições e h_class com "<< c << " posicoes"<<std::endl;

	   	t_tmp2=(unsigned long int) clock();

	    std::cout<<"  <Calculou v_blocos com "<< v_blocos <<" lins=" << lins << " threads com "<< v_threads <<std::endl;
	    std::cout<<"  <dsize "<< dsize << " colsIn="<<colsIn<<" colsOut="<< colsOut<<std::endl;
	    t_tmp3=(unsigned long int) clock();
	    hipStreamSynchronize(strm0);
	    hipStreamSynchronize(strm1);
	    t_tmp4=(unsigned long int) clock();

	    //std::cout<<"  <Sincronizou -------------------"<<std::endl;

	    fnl="log/Classif_StreamG7-T"+ s_threads +dateStr+".log.txt";
	    //arqo=fnl.c_str();
	  	std::ofstream mylog (fnl.c_str());
	  	//std::ofstream mylog (arqo);
	  	mylog<<"Processado em "<< dateStr <<std::endl;
	  	mylog<<"Processado em "<< v_blocos <<" blocos com "<< v_threads <<" threads, com "<< partes <<" partes"<<std::endl;
		mylog<<"Tempo total de classificaçao (ler CSV e classificar via stream/kernel)= "<< calcula_tempo(t_tmp1, t_tmp2) <<std::endl;
		//mylog<<"Tempo total de cópia host >> device = "<< calcula_tempo(t_tmp1, t_tmp2) <<std::endl;
		mylog<<"Tempo total de Stream Synchronize >> host = "<< calcula_tempo(t_tmp3, t_tmp4) <<std::endl;

		/*----   fecha o arquivo de entrada de registros a classificar*/
	    fin.close();

	   	/*--- cria o nome do arquivo csv de saída com as classificações ----*/
	    //fn="/home/UFF/GPU/Trabalho/Dados/Classif_Kernel"+dateStr+".csv";
	    fn="csv/Classif_StreamT"+ s_threads +dateStr+".csv";
	    //std::cout<<std::endl<<fn <<std::endl;
	    t_tmp=(unsigned long int) clock();

	    /*--- abre o csv de saída ---*/
	    std::ofstream myfile (fn.c_str());
	    myfile<<"Indice,IdClasse"<<std::endl;

	    /*---  exporta o conteúdo do vetor h_class  ---*/
	    for (i=0; i<csize; i+=colsOut)
	  	{
	  		myfile<<h_class[i]<<','<<h_class[i+1]<<"\n";
	  	}
	  	myfile.close();

		mylog<<"Tempo para exportar classificaçao para CSV= "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;

	    t_tmp=(unsigned long int) clock();

	  	/*-------------  libera memoria ------------------------*/
	  	hipFree(d_dados_0);
	  	hipFree(d_class_0);
	  	hipFree(d_dados_1);
	  	hipFree(d_class_1);
	  	hipHostFree(h_dados);
	  	hipHostFree(h_class);

		mylog<<"Tempo para liberar memoria GPU= "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;

	  	// desaloca a matriz << no Thtrust a desalocação dos vetores é transparente ---------------
		//mylog<<"Tempo para free matriz = "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;	  	
		/* tempo final */
	  	t_fin  = (unsigned long int) clock();
	  	mylog<<"Total de registros classificados= "<< lins <<std::endl;
	  	mylog<<"Tempo total de processamento= "<< setprecision(6) << calcula_tempo(t_ini, t_fin) <<std::endl;

	  	gettimeofday(&end, NULL);
    	delta = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;
    	mylog<<"Tempo total de processamento 2 = "<< delta <<std::endl;

	  	mylog.close();
	  	std::cout<<std::endl<<"Tempo total de processamento= "<< calcula_tempo(t_ini, t_fin) <<std::endl;
	  	std::cout<<"Tempo total de processamento 2 = "<< delta <<std::endl;
 	}   
 	else
 	{
 		std::cout<<std::endl<<"Erro na abertura do arquivo "<< nome <<std::endl;
 	}
}



//---------------------------------------------------------------------------
int main(int argc, char * argv[]) 
{   
  long nlin=0;
  int nthd=0;
  if (argc < 4){
    std::cout<<"Digite o nome do arquivo de entrada e a quantidade de registros e quantas threads"<<std::endl;
    abort();
  }
  // File
  std::cout<<"  <Arquivo de entrada> = "<<argv[1]<<std::endl;
   
  //nlin=std::stol(argv[2]);
  nlin=std::atol(argv[2]);
  nthd=std::atoi(argv[3]);
  /* processa a classificaçao */
  std::cout<<"  <Qtd Registros> = "<<nlin<<std::endl;
  Classif_GPU(argv[1],nlin,nthd,argv[3]); 
} 
