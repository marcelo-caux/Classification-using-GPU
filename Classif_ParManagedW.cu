
#include <hip/hip_runtime.h>
#include <iostream> 
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <stdlib.h>
#include <locale>
#include <string>
#include <limits>
#include <time.h>
#include <stdio.h>
#include <iomanip>
#include <sys/time.h>

using namespace std;

//------------ Kernel de Processamento
__global__ void Classif(int* d_dados, int* d_class, long dsize, int colsIn, int colsOut) 

{
	int i=(threadIdx.x * colsIn) + (blockIdx.x * blockDim.x * colsIn);
	int o=(threadIdx.x * colsOut) + (blockIdx.x * blockDim.x * colsOut);
	int VlOpen,VlHigh,VlLow,VlClose,classe;

	if (i<=dsize) {
		VlOpen = d_dados[i+1];
		VlHigh = d_dados[i+2];
		VlLow = d_dados[i+3];
		VlClose = d_dados[i+4];
		
		classe=(VlOpen==VlClose ? 512: VlOpen>VlClose ? 256:1024)+(VlLow<VlOpen ? 1:4)+(VlLow<VlClose ? 2:8)+(VlHigh>VlOpen ? 16:64)+(VlHigh>VlClose ? 32:128);

		d_class[o]=d_dados[i];
		d_class[o+1]=classe;
	}
}

//--------------------- Funcoes de tempo --------------------------------
std::string DataHora()
{
  time_t rawtime;
  struct tm * timeinfo;
  char buffer [20];

  time ( &rawtime );
  timeinfo = localtime ( &rawtime );
  strftime (buffer,20,"%F %H-%M-%S",timeinfo);
  return buffer;
}
/* funcao de tempo */
double calcula_tempo(const unsigned long int ini, const unsigned long int fim)
{
  double r;

  if(fim >= ini)
    r = ((double)(fim - ini)) / CLOCKS_PER_SEC;
  else
    r = ((double)( (fim + (unsigned long int)-1) - ini)) / CLOCKS_PER_SEC;
  return r;
}



//------- Classif_paralela:: / std::string ---------------------------

void Classif_GPU(const char * nome, long plins, int nthd, const char * sthd){

	char 			arq[256];
	//char 			arqo[256];
	//std::ifstream 	fin;
	int 			colsIn=5, colsOut=2;
	long 			lins,i, c, dsize, csize;
	//int 			classe,VlOpen,VlHigh,VlLow,VlClose;
	int 			v_blocos,v_threads;
	std::string		sIndice,sVlOpen,sVlHigh,sVlLow,sVlClose;
	unsigned long int 	t_ini;
	unsigned long int 	t_fin;
	unsigned long int 	t_tmp;

	std::string dateStr,fn,fnl,s_threads;
	/*--- define variaveis de tempo -------------*/
	timeval start, end;
	double delta;

	dateStr=DataHora();

	std::cout<<"  <DataHora > = "<<dateStr<<std::endl;

	/* tempo inicial */
  	t_ini  = (unsigned long int) clock;
 	gettimeofday(&start, NULL); //marcador de início do processamento

  	/* -- define as dimensões dos vetores que serão criados em logar de matrizes */
  	/* -- dsize define o tamanho do vetor de dados em função do numero de linhas e colunas*/
  	dsize=plins*colsIn;

   	/* -- csize define o tamanho do vetor de classificacao em função do numero de linhas e colunas*/
  	csize=plins*colsOut;

  	/* -- Cria os vetores que conterão os dados lidos do arquivo e a classificação */
  	int *h_dados;
  	int *h_class;

  	//std::cout<<"dsize= "<< dsize <<" csize= "<< csize<<std::endl;

  	size_t d_nbytes=dsize * sizeof(int);
  	size_t c_nbytes=csize * sizeof(int);

  	hipMallocManaged ((void**)&h_dados, d_nbytes);
  	hipMallocManaged ((void**)&h_class, c_nbytes);

  	//h_dados[0]=0;
  	//h_dados[1]=1;
	//std::cout<<"h_dados[0]= "<< h_dados[0] <<" h_dados[1]= "<< h_dados[1]<<std::endl;  	

  	lins=plins-0; 
  	std::cout<<"  <inicializou lns> = "<<lins<<std::endl;
  	/* -----  Abre o arquivo csv e inicia a carga dos vetores ------------------- */
	strcpy(arq,nome);
	ifstream fin(arq);
	if (fin.is_open()) 
	{	  	
	  	t_tmp=(unsigned long int) clock();

	  	/*---  carrega o arquivo no vetor host h_dados e inicializa h_class, transformando valores float em int*/
	  	i=0;
	  	c=0;
	  	while (fin.good())
	  	{
			getline(fin,sIndice,',');
	      	getline(fin,sVlOpen,',');
	      	getline(fin,sVlHigh,',');
	      	getline(fin,sVlLow,',');
	      	getline(fin,sVlClose,'\n');
	      	//std::cout<<"sIndice= "<< sIndice <<"sVlOpen= "<< sVlOpen<<"sVlHigh= "<< sVlHigh<<"sVlLow= "<< sVlLow<<"sVlClose= "<< sVlClose<<std::endl;
	      	//h_dados[i]=std::stoi(sIndice);
	      	h_dados[i]=std::atoi(sIndice.c_str());
	      	//h_dados[i+1]=static_cast<int>(std::stof(sVlOpen,NULL)*100);
	      	h_dados[i+1]=static_cast<int>(std::atof(sVlOpen.c_str())*100);
	      	h_dados[i+2]=static_cast<int>(std::atof(sVlHigh.c_str())*100);
	      	h_dados[i+3]=static_cast<int>(std::atof(sVlLow.c_str())*100);
	      	h_dados[i+4]=static_cast<int>(std::atof(sVlClose.c_str())*100);

	      	h_class[c]=0;
	      	h_class[c+1]=0;
	      	
	      	i+=colsIn;
	      	c+=colsOut;
	    }

	    //std::cout<<"  <Carregou h_dados com "<< i <<" posições e h_class com "<< c << " posicoes"<<std::endl;
	    /*--- Calcula o número de blocos e threads em função do número de registros 
			i = número de posições geradas para o vetor vezes o número de colunas de entrada (colsIn)
			Fixei as threads em 256
			Para processar todas as linhas do arquivo de entrada, plins, uso i/colsIN que tem o mesmo valor de plins
			assim, para 17.000.000 de registros a classificar tremos:
			v_blocos=ceil((85.000.000/5)/256)=66406,26 ==> 66407 blocos
	    ---*/

	    v_threads=nthd;
	    s_threads=std::string(sthd);
		//s_threads = "64";
	    //v_blocos=ceil((i/colsIn)/v_threads);
	    v_blocos=(int)ceil((float)lins/v_threads);
	    //std::cout<<"  <Calculou v_blocos com "<< v_blocos <<" threads com "<< v_threads <<std::endl;

	    /*--- invoca o kernel de classificação ---*/

	    Classif<<<v_blocos,v_threads>>>(h_dados, h_class, dsize, colsIn, colsOut);

	    /*--- copia de volta o vetor de classicação --*/

	    hipDeviceSynchronize();
	    //std::cout<<"  <Sincronizou -------------------"<<std::endl;

	    fnl="log/Classif_KernelT"+ s_threads +dateStr+".log.txt";
	    //arqo=fnl.c_str();
	  	std::ofstream mylog (fnl.c_str());
	  	//std::ofstream mylog (arqo);
	  	mylog<<"Processado em "<< dateStr <<std::endl;
	  	mylog<<"Processado em "<< v_blocos <<" blocos com "<< v_threads <<" threads"<<std::endl;
		mylog<<"Tempo total de classificaçao (ler CSV e classificar via kernel)= "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;

		/*----   fecha o arquivo de entrada de registros a classificar*/
	    fin.close();
	   	//mylog<<"Tempo decorrido até o final da classificaçao= "<< calcula_tempo(t_ini, (unsigned long int) clock()) <<std::endl;

	   	/*--- cria o nome do arquivo csv de saída com as classificações ----*/
	    //fn="/home/UFF/GPU/Trabalho/Dados/Classif_Kernel"+dateStr+".csv";
	    fn="csv/Classif_KernelT"+ s_threads +dateStr+".csv";
	    //std::cout<<std::endl<<fn <<std::endl;
	    t_tmp=(unsigned long int) clock();

	    /*--- abre o csv de saída ---*/
	    std::ofstream myfile (fn.c_str());
	    myfile<<"Indice,IdClasse"<<std::endl;

	    /*---  exporta o conteúdo do vetor h_class  ---*/
	    for (i=0; i<csize; i+=colsOut)
	  	{
	  		myfile<<h_class[i]<<','<<h_class[i+1]<<"\n";
	  	}
	  	myfile.close();
		mylog<<"Tempo para exportar classificaçao para CSV= "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;

	  	// desaloca a matriz << no Thtrust a desalocação dos vetores é transparente ---------------
		//mylog<<"Tempo para free matriz = "<< calcula_tempo(t_tmp, (unsigned long int) clock()) <<std::endl;	  	
		/* tempo final */
	  	t_fin  = (unsigned long int) clock();
	  	mylog<<"Total de registros classificados= "<< lins <<std::endl;
	  	mylog<<"Tempo total de processamento= "<< setprecision(6) << calcula_tempo(t_ini, t_fin) <<std::endl;

	  	gettimeofday(&end, NULL);
    	delta = ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;
    	mylog<<"Tempo total de processamento 2 = "<< delta <<std::endl;

	  	mylog.close();
	  	std::cout<<std::endl<<"Tempo total de processamento= "<< calcula_tempo(t_ini, t_fin) <<std::endl;
	  	std::cout<<"Tempo total de processamento 2 = "<< delta <<std::endl;
 	}   
 	else
 	{
 		std::cout<<std::endl<<"Erro na abertura do arquivo "<< nome <<std::endl;
 	}
}



//---------------------------------------------------------------------------
int main(int argc, char * argv[]) 
{   
  long nlin=0;
  int nthd=0;
  if (argc < 4){
    std::cout<<"Digite o nome do arquivo de entrada e a quantidade de registros e quantas threads"<<std::endl;
    abort();
  }
  // File
  std::cout<<"  <Arquivo de entrada> = "<<argv[1]<<std::endl;
   
  //nlin=std::stol(argv[2]);
  nlin=std::atol(argv[2]);
  nthd=std::atoi(argv[3]);
  /* processa a classificaçao */
  std::cout<<"  <Qtd Registros> = "<<nlin<<std::endl;
  Classif_GPU(argv[1],nlin,nthd,argv[3]); 
} 
